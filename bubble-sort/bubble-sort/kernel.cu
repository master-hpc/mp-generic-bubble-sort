#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void sortKernel(int *v, int n)
{
	for (int i = 0; i < n; i++) {

		int indice = 2 * threadIdx.x + (i % 2);

		if ((indice + 1 < n) && (v[indice] > v[indice + 1])) {
			int m = v[indice];
			v[indice] = v[indice + 1];
			v[indice + 1] = m;
		}

	}
}

int main()
{
	const int arraySize = 10;
	int v[arraySize] = { 3, 7, 1, 10, 6, 9, 5, 2, 8, 4 };


	int *dev_v = 0;

	hipMalloc((void**)&dev_v, arraySize * sizeof(int));

	hipMemcpy(dev_v, v, arraySize * sizeof(int), hipMemcpyHostToDevice);

	sortKernel << <1, arraySize / 2 >> >(dev_v, arraySize);

	hipMemcpy(v, dev_v, arraySize * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_v);

	for (int i = 0; i < arraySize; i++) {
		printf(" %d ", v[i]);
	}

	return 0;
}

