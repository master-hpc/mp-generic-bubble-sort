
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void bubbleSort(int *v, const int n);

int main(int argc, char** argv)
{
	// vector size
	const int size = 10;

	// host vector
	int h_v[size] = { 3, 7, 1, 10, 6, 9, 5, 2, 8, 4 };

	// device vector
	int *d_v = 0;

	hipMalloc((void**)&d_v, size * sizeof(int));

	hipMemcpy(d_v, h_v, size * sizeof(int), hipMemcpyHostToDevice);

	dim3 grdDim(1, 1, 1);
	dim3 blkDim(size / 2, 1, 1);

	bubbleSort <<<grdDim, blkDim>>>(d_v, size);

	hipMemcpy(h_v, d_v, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_v);

	for (int i = 0; i < size; i++) {
		std::cout << (i == 0 ? "{ " : "") <<  h_v[i] << (i < size -1 ? " ," : "}");
	}
	std::cout << std::endl;
	return 0;
}

__global__ void bubbleSort(int *v, const int n)
{
	int tIdx = threadIdx.x;

        for (int i = 0; i < n; i++) {

		int offset = i % 2;
                int indiceGauche = 2 * tIdx + offset;
		int indiceDroite = indiceGauche + 1;

                if (indiceDroite < n) {
			if (v[indiceGauche] > v[indiceDroite]) {
				int tmp = v[indiceGauche];
                        	v[indiceGauche] = v[indiceDroite];
                        	v[indiceDroite] = tmp;
			}
                }
		__syncthreads();

        }
}
